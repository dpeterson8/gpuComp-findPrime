#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
extern "C" {
  #include "timing.h"
}

typedef unsigned long long bignum;

__host__ int isPrime(bignum num);
__device__ int disPrime(bignum num);
void checkPrimes(char * result, bignum num, bignum n);
int totalPrimes(char * arr, int size);
__global__ void dCheckPrimes(char * result);

int main(int argc, char *argv[]) {
  // doubles used for time comp
  double now, then;
  double scost, pcost;

  bignum *h_n, *h_s =  (bignum *) malloc(sizeof(bignum));
  h_n = (bignum *) malloc(sizeof(bignum));
  *h_n = atoi(argv[1]);
  bignum blockSize;
  blockSize = atoi(argv[2]);
  char * result = (char *) malloc((*h_n + 1) *sizeof(int));
  char * d_result;

  // find primes using cpu and measure time below
  then = currentTime();
  checkPrimes(result, 0, *h_n);
  now = currentTime();
  scost = (now - then) * 1000;
  printf("time taken calculating primes using cpu: %lf ms\n", scost);
  int tempPrime = totalPrimes(result, (*h_n + 1));
  printf("Total primes found: %d\n", tempPrime);

  // find primes using gpu and measure time below
  then = currentTime();
  hipMalloc((void**) &d_result, *h_n * sizeof(int));
  hipMemcpy( d_result, result, *h_n * sizeof(int), hipMemcpyHostToDevice);
  dCheckPrimes<<<ceil((*h_n+1)/2.0/blockSize),blockSize>>>( d_result);
  hipMemcpy( result, d_result, *h_n * sizeof(int), hipMemcpyDeviceToHost);
  now = currentTime();
  pcost = (now - then) * 1000;
  printf("time taken calculating primes using cuda: %lf ms\n", pcost);
  tempPrime = totalPrimes(result, (*h_n + 1));
  printf("Total primes found: %d\n", tempPrime);

  //free used memory
  hipFree(d_result);
  free(result);
  free(h_n);
  free(h_s);
}

/*
  dCheckPrimes: kernal function used to start operations on the gpu using cuda

  result -> the array that the function will return the results too
*/
__global__ void dCheckPrimes(char * result) {
  // get thread id which will also map to postion in array
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // check if current value is 0 if so bump id to 2 (first prime)
  if (id == 0) { id += 2;}
  else { 
    // will cause threads to skip even positions in array
    id = id + id + 1;
  }
  result[id] = disPrime(id);
}

/*
  checkPrimes: function used to find primes in an array using the cpu

  result -> the array that the function will return the results too
  n -> number to check for prime up too
*/
void checkPrimes(char * result, bignum num, bignum n) {
  bignum i;

  // if num is even add 1 to make odd
  if(num % 2 == 0) { num++; }

  // go thorugh odd numbers checking if each is prime
  for(i=num; i<num+n; i = i+2) {
    result[i] = isPrime(i);
  }

}

/*
  isPrime: takes one integer and checks if integer is prime or not used by host

  num -> integer to check if prime
*/
__host__ int isPrime(bignum num) {
  
  bignum i;
  bignum lim = (bignum) sqrt(num) + 1;

  for(i = 2; i<lim; i++) {
    if(num % i == 0) {
      return 0;
    }
  }
  return 1;
}

/*
  disPrime: takes one integer and checks if integer is prime or not used by device

  num -> integer to check if prime
*/
__device__ int disPrime(bignum num) {

  bignum i;
  bignum lim = (bignum) sqrtf(num) + 1;

  for(i = 2; i<lim; i++) {
    if(num % i == 0) {
      return 0;
    }
  }
  return 1;
}

/*
  totalPrimes: function used to check for total amount of primes in array

  arr -> array filled with primes to count
  size -> size of array being passed in 
*/
int totalPrimes(char * arr, int size) {
  int j = 0;
  for(int i = 0; i < size; i++) {
    if(arr[i] == 1) {
      j++;
    }
  }

  return j;
}